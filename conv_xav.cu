#include "hip/hip_runtime.h"
//
// CUDA implementation of Laplacian Filter
//
//#include "opencv2/opencv.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
//#include <qdebug.h>
#include "main.h"



using namespace std;

const int inputSize = sizeof(__half)*1920 * 1080;
const int outputSize = sizeof(__half)*1920 * 1080;
const int kernelSize = sizeof(__half)*FILTER_WIDTH * FILTER_HEIGHT;
__half *d_input, *d_output;
__half *d_kernel;


bool initialized=false;

// Run Laplacian Filter on GPU
__global__ void laplacianFilter(__half*srcImage, __half*dstImage, unsigned int width, unsigned int height, __half* kernel0)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

   //float kernel[3][3] = {-1, -1, -1, -1, 8, -1, -1, -1, -1};
   // only threads inside image will write results
   if((x>=FILTER_WIDTH/2) && (x<(width-FILTER_WIDTH/2)) && (y>=FILTER_HEIGHT/2) && (y<(height-FILTER_HEIGHT/2)))
   {
         // Sum of pixel values
         __half sum = 0.0;
         // Loop inside the filter to average pixel values
         for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++) {
            for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++) {
               __half fl = srcImage[((y+ky)*width + (x+kx))];
               __half a = __hmul(fl , kernel0[(ky + FILTER_HEIGHT / 2) * FILTER_WIDTH + kx + FILTER_WIDTH / 2]);
              /* sum += fl*kernel0[(ky+FILTER_HEIGHT/2)*FILTER_WIDTH + kx+FILTER_WIDTH/2];*/
               sum = __hadd(sum, a);
            }
         }
         dstImage[(y*width+x)] =  sum;
   }
}


// The wrapper to call laplacian filter
extern "C" void laplacianFilter_GPU_wrapper(const cv::Mat& input, cv::Mat& output, const cv::Mat& kernel)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        if (!initialized)
        {
            cout<<"111"<<endl;
            // Allocate device memory
            hipMalloc(&d_input,inputSize);
            hipMalloc(&d_output,outputSize);
            hipMalloc(&d_kernel,kernelSize);

            initialized=true;
        }

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyHostToDevice);
//        cout<<"22222"<<endl;
        hipMemcpy(d_kernel,kernel.ptr(),kernelSize,hipMemcpyHostToDevice);
//        cout <<"      "<<kernel<<endl;

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start,0);

        laplacianFilter<<<grid,block>>>((__half*)d_input, (__half*)d_output, output.cols, output.rows, (__half*)d_kernel);

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        printf("Kernel time: %.4f ms\n", milliseconds);

        //Copy data from device memory to output image

        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToHost);
//        cout <<output<<endl;

        //Free the device memory
//        hipFree(d_input);
//        hipFree(d_output);
//        hipFree(d_kernel);


}


/////////////////////////////////////////////////////////////////////copy////////////////////////////////////////////////////////////////////////


const int indataLength = sizeof(__half) * IMG_HEIGHT * IMG_WIDTH;
const int outdataLength = sizeof(__half) * IMG_HEIGHT * IMG_WIDTH;
const int kerneldataLength = sizeof(__half) * FILTER_WIDTH * FILTER_HEIGHT;
__half* d_indata, * d_outdata; //指针类型虽然不影响，内存分配大小，但是严谨使用，应该与目标数据的类型（fp16）一至
__half* d_kerneldata;

bool first = true;

__global__ void convfp16_kernel(const __half* srcImage, __half* dstImage, const __half* kernel,const unsigned int width, const unsigned int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;  //block在grid中的序号 block在x维度的大小 线程在一个block中的序号
    int y = blockIdx.y * blockDim.y + threadIdx.y;   //线程的全局index

       // only threads inside image will write results
    if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
    {
        // Sum of pixel values
        __half sum = 0.0;
        // Loop inside the filter to average pixel values
        for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++)
        {
            for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++)
            {
                __half a = __hmul(srcImage[((y + ky) * width + (x + kx))],\
                    kernel[(ky + FILTER_HEIGHT / 2) * FILTER_WIDTH + kx + FILTER_WIDTH / 2]);
                sum = __hadd(sum, a);
            }
        }
        dstImage[(y * width + x)] = sum;
    }
//    else if (x < width && y < height)
//    {
//        //dstImage[(y * width + x)] = srcImage[(y * width + x)]*0.3;
//        dstImage[(y * width + x)] = __hmul(srcImage[(y * width + x)], 0.4); //cuda fp16一定要用自己的加减乘除
//    }

}

extern "C" void convfp16(const __half* indata, __half* outdata, const __half* kerneldata, const int width, const int height)
{

    // Use cuda event to catch time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (first)
    {
        // Allocate device memory
        //hipMalloc<unsigned char>(&d_input, inputSize);
        //hipMalloc<unsigned char>(&d_output, outputSize);
        //hipMalloc<float>(&d_kernel, kernelSize);
        //反复在GPU上开辟内存会浪费大量时间
        hipMalloc((void**)&d_indata, indataLength);
        hipMalloc((void**)&d_outdata, outdataLength);
        hipMalloc((void**)&d_kerneldata, kerneldataLength);
        first = false;
    }

    // Copy data from OpenCV input image to device memory
    hipMemcpy(d_indata, indata, indataLength, hipMemcpyHostToDevice);
    hipMemcpy(d_kerneldata, kerneldata, kerneldataLength, hipMemcpyHostToDevice);

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    //Start time
    hipEventRecord(start, 0);

    convfp16_kernel << <grid, block >> > (d_indata, d_outdata, d_kerneldata, width, height); //右值直接赋值

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Kernel time: %.2f ms\n", milliseconds);

    //Copy data from device memory to output image
    hipMemcpy(outdata, d_outdata, outdataLength, hipMemcpyDeviceToHost);

    //Mat indataMat =Mat(height, width, CV_16FC1, (uchar*)indata); //还识别不了__half*,const应该都可以做右值
    //cout << indataMat(Rect(10, 10, 10, 10));


    //Free the device memory
//        hipFree(d_input);
//        hipFree(d_output);
//        hipFree(d_kernel);


}


/////////////////////////////////////////////////////////////////////copy////////////////////////////////////////////////////////////////////////

/////////////////////////////////////////////////////////////////////after vs debug cufft mulSpectrum////////////////////////////////////////////////////////////////////////

/*
核函数在访问内存时会自动结束函数
频谱相乘之后的逆变换的图像会被截取到不同的位置
*/



//Nvidia cufft https://docs.nvidia.com/cuda/cufft/index.html#cufft-setup
/// <summary>
/// 对核函数傅立叶变换，kernel  real (M,N)->complex (M,N/2+1)
///
/// 2D	C2C	  (N1,N2)     hipfftComplex	(N1,N2)     hipfftComplex
/// 2D	C2R	  (N1,N2/2+1) hipfftComplex	(N1,N2)     hipfftReal
/// 2D	R2C	  (N1,N2)     hipfftReal	    (N1,N2/2+1) hipfftComplex
///  N1,nx <=> h; N2,ny <=> w
///
/// </summary>
/// <param name="indata"></变换之前的实数数据，h*w*sizeof(hipfftReal) bytes>
/// <param name="outdata"></变换之后的复数数据，h*(w/2+1)*sizeof(hipfftComplex) bytes>
/// <param name="kh"></ height of image(or kernel),both have the same size>
/// <param name="kw"></weight of image(or kernel)>

void fftKernel(hipfftReal* indata, hipfftComplex* outdata, const unsigned int kh, const unsigned int kw)
{
    //imgh == kh
    //imgw == kw
    hipfftReal* indata_dev;
    hipfftComplex* outdata_dev;
    hipMalloc((void**)&indata_dev, sizeof(hipfftReal) * kh * kw);
    hipMalloc((void**)&outdata_dev, sizeof(hipfftComplex) * kh * (kw / 2 + 1));
    hipMemcpy(indata_dev, indata, sizeof(hipfftReal) * kw * kh, hipMemcpyHostToDevice);

    hipfftHandle planForward;
    hipfftPlan2d(&planForward, kh, kw, HIPFFT_R2C);
    hipfftExecR2C(planForward, indata_dev, outdata_dev);
    hipDeviceSynchronize();

    hipMemcpy(outdata, outdata_dev, sizeof(hipfftComplex) * kh * (kw / 2 + 1), hipMemcpyDeviceToHost);

    hipfftDestroy(planForward);
    hipFree(outdata_dev);
    hipFree(indata_dev);
}


/// <summary>
/// 频谱相乘的核函数，卷积运算 互相关运算
/// </summary>
/// <param name="imgdata_dev"></在设备上的图像数据>
/// <param name="outdata_dev"></必须out-of-place存储计算后的数据，否则会有污染>
/// <param name="kdata_dev"></卷积核模板，可以始终放在设备上>
/// <param name="imgh"></height of image>
/// <param name="imgw"></width of image>
/// <returns></returns>

__global__ void kernel_convfft(const hipfftComplex* imgdata_dev, hipfftComplex* outdata_dev, const hipfftComplex* kdata_dev, const unsigned int imgh, const unsigned int imgw)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //    if(x< imgw && y < imgh/2 )
    //    {
    //        imgdata_dev[y*imgw + x].x = imgdata_dev[y*imgw + x].x * kdata_dev[y*imgw + x].x;
    //        imgdata_dev[y*imgw + x].y = imgdata_dev[y*imgw + x].y * kdata_dev[y*imgw + x].y;
    //    }
    //    else if(y >= imgh/2 && x <imgw && y <imgh)
    //    {
    //        imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].x = imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].x * kdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].x;
    //        imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].y = imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].y * kdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].y;
    //    }
    //if (x == 0 || y == 0)
    //{
    //	imgdata_dev[y * imgw + x].x = imgdata_dev[y * imgw + x].x * kdata_dev[y * imgw + x].x - imgdata_dev[y * imgw + x].y * kdata_dev[y * imgw + x].y;
    //	imgdata_dev[y * imgw + x].y = imgdata_dev[y * imgw + x].y * kdata_dev[y * imgw + x].x + imgdata_dev[y * imgw + x].x * kdata_dev[y * imgw + x].y;
    //}
    //else if (y > 0 && x > 0 && x < (imgw / 2 + 1))
    //{
    //	imgdata_dev[y * imgw + x].x = imgdata_dev[y * imgw + x].x * kdata_dev[y * imgw + x].x - imgdata_dev[y * imgw + x].y * kdata_dev[y * imgw + x].y;
    //	imgdata_dev[y * imgw + x].y = imgdata_dev[y * imgw + x].y * kdata_dev[y * imgw + x].x + imgdata_dev[y * imgw + x].x * kdata_dev[y * imgw + x].y;
    //}
    unsigned int h = imgh;
    unsigned int w = imgw / 2 + 1;
    if (x < w && y < h) //设备上访问非法内存的线程会被停止
    {
        //float ri = imgdata_dev[y * w + x].x;
        //float ii = imgdata_dev[y * w + x].y;
        //float rk = kdata_dev[y * w + x].x;
        //float ik = kdata_dev[y * w + x].y;  //核的虚数部位

        //if ()
        //{
            outdata_dev[y * w + x].x = imgdata_dev[y * w + x].x * kdata_dev[y * w + x].x - imgdata_dev[y * w + x].y * kdata_dev[y * w + x].y; //原来写成imgw
            outdata_dev[y * w + x].y = imgdata_dev[y * w + x].y * kdata_dev[y * w + x].x + imgdata_dev[y * w + x].x * kdata_dev[y * w + x].y;
        //}
        //else if (x == 0 && y == 0)
        //{
        //	imgdata_dev[y * w + x].x = imgdata_dev[y * w + x].x * kdata_dev[y * w + x].x;
        //}
    }
}

bool first1 = true;
hipfftReal* imgdata_dev;
hipfftReal* outdata_dev;
hipfftComplex* outdataComplex_dev;
hipfftComplex* imgdataComplex_dev;
hipfftComplex* kdata_dev;
hipfftHandle planForward, planInverse;

void fftImgKernel(const hipfftReal* imgdata, hipfftReal* outdata, const hipfftComplex* kdata, const unsigned int imgh, const unsigned int imgw)
{
    if (first1)
    {
        //img
        hipMalloc((void**)&imgdata_dev, sizeof(hipfftReal) * imgh * imgw);
        hipMalloc((void**)&outdata_dev, sizeof(hipfftReal) * imgh * imgw);
        //img complex
        hipMalloc((void**)&imgdataComplex_dev, sizeof(hipfftComplex) * imgh * (imgw / 2 + 1));//数据要对齐，前面可能就是数据错误
        hipMalloc((void**)&outdataComplex_dev, sizeof(hipfftComplex) * imgh * (imgw / 2 + 1));
        //kernel
        hipMalloc((void**)&kdata_dev, sizeof(hipfftComplex) * imgh * (imgw / 2 + 1));
        //handle
        hipfftPlan2d(&planForward, imgh, imgw, HIPFFT_R2C);
        hipfftPlan2d(&planInverse, imgh, imgw, HIPFFT_C2R);
        first1 = false;
    }
    //memory copy kernel and image from host to device
    hipMemcpy(kdata_dev, kdata, sizeof(hipfftComplex) * imgh * (imgw / 2 + 1), hipMemcpyHostToDevice);
    hipMemcpy(imgdata_dev, imgdata, sizeof(hipfftReal) * imgh * imgw , hipMemcpyHostToDevice);
    //fft forward transform
    hipfftExecR2C(planForward, imgdata_dev, imgdataComplex_dev);
    hipDeviceSynchronize();
    //multiply kernel and image
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((imgw/2 + 1 + block.x - 1) / block.x, (imgh + block.y - 1) / block.y);
    kernel_convfft << <grid, block >> > (imgdataComplex_dev, outdataComplex_dev, kdata_dev, imgh, imgw);
    //fft inverse transform
    hipfftExecC2R(planInverse, outdataComplex_dev, outdata_dev);
    hipDeviceSynchronize();
    hipMemcpy(outdata, outdata_dev, sizeof(hipfftReal) * imgh * imgw, hipMemcpyDeviceToHost);
    //Mat outImg = Mat(imgh, imgw, CV_32F, outdata);
    //cout << outImg(Rect(0, 0, 15, 15)) << endl;
    //    hipFree(imgdata_dev);
    //    hipFree(imgdataComplex_dev);
    //    hipFree(kdata_dev);
    //    hipfftDestroy(planForward);
    //    hipfftDestroy(planInverse);
}

/////////////////////////////////////////////////////////////////////after vs debug cufft mulSpectrum///////////////////////////////////////////////////////
