#include "hip/hip_runtime.h"
//
// CUDA implementation of Laplacian Filter
//

#include "main.h"
//#include <qdebug.h>

#define BLOCK_SIZE      8
#define FILTER_WIDTH    85
#define FILTER_HEIGHT   85

using namespace std;

const int inputSize = sizeof(uchar) * 1920 * 1080;
const int outputSize = sizeof(uchar) * 1920 * 1080;
const int kernelSize = sizeof(float) * FILTER_WIDTH * FILTER_HEIGHT;
unsigned char* d_input, * d_output;
float* d_kernel;


bool initialized = false;

// Run Laplacian Filter on GPU
__global__ void laplacianFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height, float* kernel0)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;  //block��grid�е���� block��xά�ȵĴ�С �߳���һ��block�е����
    int y = blockIdx.y * blockDim.y + threadIdx.y;   //�̵߳�ȫ��index

    //float kernel[3][3] = {-1, -1, -1, -1, 8, -1, -1, -1, -1};
    // only threads inside image will write results
    if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
    {
        // Sum of pixel values
        float sum = 0;
        // Loop inside the filter to average pixel values
        for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
            for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
                float fl = srcImage[((y + ky) * width + (x + kx))];
                sum += fl * kernel0[(ky + FILTER_HEIGHT / 2) * FILTER_WIDTH + kx + FILTER_WIDTH / 2];
            }
        }
        dstImage[(y * width + x)] = sum;
    }
}


// The wrapper to call laplacian filter
extern "C" void laplacianFilter_GPU_wrapper(const cv::Mat & input, cv::Mat & output, const cv::Mat & kernel)
{
    // Use cuda event to catch time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (!initialized)
    {
        cout << "111" << endl;
        // Allocate device memory
        hipMalloc<unsigned char>(&d_input, inputSize);
        hipMalloc<unsigned char>(&d_output, outputSize);
        hipMalloc<float>(&d_kernel, kernelSize);

        initialized = true;
    }

    // Copy data from OpenCV input image to device memory
    hipMemcpy(d_input, input.ptr(), inputSize, hipMemcpyHostToDevice);
    //        cout<<"22222"<<endl;
    hipMemcpy(d_kernel, kernel.ptr(), kernelSize, hipMemcpyHostToDevice);

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Start time
    hipEventRecord(start, 0);

    laplacianFilter <<<grid, block >>> (d_input, d_output, output.cols, output.rows, d_kernel);  //�������߳̿��ά�ȣ�һ���ж��ٵ��߳̿飻�߳̿����̵߳�ά�ȣ�ÿ���߳̿�һ���ж��ٸ��߳�

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Kernel time: %.2f ms\n", milliseconds);

    //Copy data from device memory to output image

    hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);

    //Free the device memory
//        hipFree(d_input);
//        hipFree(d_output);
//        hipFree(d_kernel);


}


