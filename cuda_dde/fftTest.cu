#include "main.h"


void fft1d()
{
	const int signalLength = 10;
	hipfftComplex* signal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * signalLength);
	for (int i = 0; i < 10; i++)
	{
		(signal+i)->x = (float)(i + 1);
		(signal+i)->y = 0;
	}
	hipfftComplex* signalDev;
	hipMalloc((void**)&signalDev, sizeof(hipfftComplex) * signalLength);
	hipMemcpy(signalDev, signal, sizeof(hipfftComplex) * signalLength, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan1d(&plan, signalLength, HIPFFT_C2C,1);
	hipfftExecC2C(plan, (hipfftComplex*)signalDev, (hipfftComplex*)signalDev, HIPFFT_FORWARD);
	hipDeviceSynchronize();//wait to be done
	hipMemcpy(signal, signalDev, signalLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++)
	{
		cout << (signal+i)->x << " " << (signal+i)->y << endl;
	}

	hipfftDestroy(plan);
	free(signal);
	hipFree(signalDev);




}