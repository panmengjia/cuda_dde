#include "hip/hip_runtime.h"
#include "main.h"

//c2c c2r r2c https://blog.csdn.net/qq_17239003/article/details/79090803
//�ܺõĲ������£�https://zhuanlan.zhihu.com/p/34587739


__global__ void kernel_normalize1d(hipfftComplex* signal,const int signalLength)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < signalLength) //�Ǳ�����ʷǷ��ڴ���
	{
		(signal + idx)->x /= signalLength;
		(signal + idx)->y /= signalLength;
	}
}

void fft1d()
{
	// Use cuda event to catch time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const int signalLength = 15;
	hipfftComplex* signal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * signalLength);
	for (int i = 0; i < signalLength; i++)
	{
		(signal+i)->x = (float)(i + 1);
		(signal+i)->y = 0;
	}
	hipfftComplex* signalDev;
	hipMalloc((void**)&signalDev, sizeof(hipfftComplex) * signalLength);
	hipMemcpy(signalDev, signal, sizeof(hipfftComplex) * signalLength, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan1d(&plan, signalLength, HIPFFT_C2C,1);
	hipfftExecC2C(plan, (hipfftComplex*)signalDev, (hipfftComplex*)signalDev, HIPFFT_FORWARD);
	hipDeviceSynchronize();//wait to be done
	hipMemcpy(signal, signalDev, signalLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < signalLength; i++)
	{
		cout << (signal+i)->x << " " << (signal+i)->y << endl;
	}

	//��任
	hipfftComplex* signalInverse = (hipfftComplex*)malloc(sizeof(hipfftComplex) * signalLength);
	hipfftComplex* signalInvDev;
	hipMalloc((void**)&signalInvDev, sizeof(hipfftComplex) * signalLength);
	for (int i = 0; i < signalLength; i++)  //ȫ����ʼ��Ϊ0������y in-place������д������ݣ�Ҳ���Բ�����y�е�����
	{
		(signal + i)->x = 0;
		(signal + i)->y = 0;
	}
	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}


	hipfftHandle planInv;
	hipfftPlan1d(&plan, signalLength, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, (hipfftComplex*)signalDev, signalInvDev, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	hipMemcpy(signalInverse, signalInvDev, signalLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}
	//�˹�һ������
	//���Բ������йܷ�ʽ
	const unsigned int blockX = 32;
	// Specify block size
	const dim3 block(blockX, 1);
	const dim3 grid(1);


	// Start time
	hipEventRecord(start, 0);  //��ʱ����
	kernel_normalize1d <<< grid, block >>> (signalInvDev,signalLength);//����ʵ���������߳�����32
	hipMemcpy(signalInverse, signalInvDev, signalLength * sizeof(hipfftComplex),hipMemcpyDeviceToHost);


	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}

	hipfftDestroy(plan);
	free(signal);
	hipFree(signalDev);

}

void fft2d()
{


}