#include "hip/hip_runtime.h"
#include "main.h"

//c2c c2r r2c https://blog.csdn.net/qq_17239003/article/details/79090803
//�ܺõĲ������£�https://zhuanlan.zhihu.com/p/34587739


__global__ void kernel_normalize1d(hipfftComplex* signal,const int signalLength)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < signalLength) //�Ǳ�����ʷǷ��ڴ���
	{
		(signal + idx)->x /= signalLength;
		(signal + idx)->y /= signalLength;
	}
}

void fft1d()
{
	// Use cuda event to catch time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const int signalLength = 15;
	hipfftComplex* signal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * signalLength);
	for (int i = 0; i < signalLength; i++)
	{
		(signal+i)->x = (float)(i + 1);
		(signal+i)->y = 0;
	}
	hipfftComplex* signalDev;
	hipMalloc((void**)&signalDev, sizeof(hipfftComplex) * signalLength);
	hipMemcpy(signalDev, signal, sizeof(hipfftComplex) * signalLength, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan1d(&plan, signalLength, HIPFFT_C2C,1);
	hipfftExecC2C(plan, (hipfftComplex*)signalDev, (hipfftComplex*)signalDev, HIPFFT_FORWARD);
	hipDeviceSynchronize();//wait to be done
	hipMemcpy(signal, signalDev, signalLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < signalLength; i++)
	{
		cout << (signal+i)->x << " " << (signal+i)->y << endl;
	}

	//��任
	hipfftComplex* signalInverse = (hipfftComplex*)malloc(sizeof(hipfftComplex) * signalLength);
	hipfftComplex* signalInvDev;
	hipMalloc((void**)&signalInvDev, sizeof(hipfftComplex) * signalLength);
	for (int i = 0; i < signalLength; i++)  //ȫ����ʼ��Ϊ0������y in-place������д������ݣ�Ҳ���Բ�����y�е�����
	{
		(signal + i)->x = 0;
		(signal + i)->y = 0;
	}
	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}


	hipfftHandle planInv;
	hipfftPlan1d(&plan, signalLength, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, (hipfftComplex*)signalDev, signalInvDev, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	hipMemcpy(signalInverse, signalInvDev, signalLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}
	//�˹�һ������
	//���Բ������йܷ�ʽ
	const unsigned int blockX = 32;
	// Specify block size
	const dim3 block(blockX, 1);
	const dim3 grid(1);


	// Start time
	hipEventRecord(start, 0);  //��ʱ����
	kernel_normalize1d <<< grid, block >>> (signalInvDev,signalLength);//����ʵ���������߳�����32
	hipMemcpy(signalInverse, signalInvDev, signalLength * sizeof(hipfftComplex),hipMemcpyDeviceToHost);


	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}

	hipfftDestroy(plan);
	free(signal);
	hipFree(signalDev);

}


vector<vector<float>> extractConvMatfft()
{
	const string& str = "C:/Users/b515/Desktop/pmj/cuda_dde/fenglin/15/";
	vector<vector<float>> HVSFT;
	HVSFT.resize(15);
	//    unsigned int counter = 0;
	for (int i = 1; i < 16; i++)
	{
		ifstream dataFile(str + to_string(i) + ".txt");
		float dataElement;
		while (dataFile >> dataElement)
		{
			HVSFT[i - 1].push_back(dataElement);
		}
		dataFile.close();
	}
	return HVSFT;
}

__global__ void kernel_normalize2d()
{

}

//�ù��̶�������ά����������
//����ָ����Բ���
void fft2d()
{
	vector<vector<float>> dataArray = extractConvMatfft();
	//for (int i = 0;  i < 15 * 15; i++)
	//{
	//	cout << *(dataArray[0].data() +i) << "  ";  //dataArray[0][i]  *(dataArray[0].data() +i)��Ȼ����Ҫǿ������ת��
	//}
	const unsigned int matSize = 20;
	hipfftReal (*mat)[5];
	mat = (hipfftReal(*)[5])malloc(sizeof(hipfftReal) * matSize);  //4row 5col
	for (int i = 0; i < 20; i++)
	{
		//*(*(mat + i/5)+i%5) = i + 1;
		*(*mat + i) = i + 1; //��ָ������϶�������е�ַ��ָ�������ֵ���ǵ�ַ
		//cout << mat[i / 5][i % 5] << endl;
	}
	hipfftReal(*mat_dev)[5];
	mat_dev = NULL;
	hipMalloc((void**)mat_dev, sizeof(hipfftReal [5]) * matSize/5); //hipfftReal
	hipMemcpy(*mat_dev, *mat, sizeof(hipfftReal) * matSize, hipMemcpyHostToDevice);
	hipfftComplex(*matForward)[5];
	matForward = (hipfftComplex(*)[5])malloc(sizeof(hipfftReal [5]) * matSize/5);  //�Ե�ַ����ǿ������ת����Ӧ�ö��Ƕ���ֵǿ������ת��
	hipfftComplex(*matForward_dev)[5];
	matForward_dev = NULL;
	hipMalloc((void**)matForward_dev, sizeof(hipfftComplex) * matSize);
	//matForward_dev = (hipfftComplex(*)[5])malloc(sizeof(hipfftComplex) * matSize);  //�Ե�ַ����ǿ������ת����Ӧ�ö��Ƕ���ֵǿ������ת��


	hipfftHandle plan;
	/*
	��άFFT�㷨ʵ���У�ͬһάFFT��ͬ���ǣ�
    (1) ���������û��BATCH��������NY��NXΪ������NYΪ������
    (2) FFT�����任������λ�ú����λ�ò�ͬ��*/
	hipfftPlan2d(&plan, 4, 5, HIPFFT_R2C);  //  �� ��
	hipfftExecR2C(plan,(hipfftReal*)*mat_dev, (hipfftComplex*)*matForward_dev);
	hipMemcpy((hipfftComplex*)*matForward, (hipfftComplex*)*matForward_dev, sizeof(hipfftComplex) * matSize, hipMemcpyDeviceToHost);
	for (int i = 0; i < matSize; ++i)
	{
		cout << (*matForward + i)->x << " " << (*matForward + i)->y << endl;
	}


	hipFree(*mat_dev);
	hipFree(*matForward_dev);
	free(*mat);
	
}

void fft3d()
{
	vector<vector<float>> dataArray = extractConvMatfft();
	//for (int i = 0;  i < 15 * 15; i++)
	//{
	//	cout << *(dataArray[0].data() +i) << "  ";  //dataArray[0][i]  *(dataArray[0].data() +i)��Ȼ����Ҫǿ������ת��
	//}
	const unsigned int matSize = 16;
	hipfftComplex* mat;
	mat = (hipfftComplex*)malloc(sizeof(hipfftComplex) * matSize);  //5row 4col
	for (int i = 0; i < matSize; i++)
	{
		(*(mat + i)).x = (i + 1); //��ָ������϶�������е�ַ��ָ�������ֵ���ǵ�ַ
		(mat + i)->y = 0;
		cout << mat[i].x << endl;
	}
	hipfftComplex*  mat_dev;  //N
	mat_dev = NULL;
	hipMalloc((void**)&mat_dev, sizeof(hipfftComplex) * matSize);  // //hipfftComplex
	hipMemcpy(mat_dev, mat, sizeof(hipfftComplex) * matSize, hipMemcpyHostToDevice); 
	hipfftComplex* matForward; //R2C N->N/2+1
	matForward = (hipfftComplex*)malloc(sizeof(hipfftComplex) * matSize);  //�Ե�ַ����ǿ������ת����Ӧ�ö��Ƕ���ֵǿ������ת��
	hipfftComplex* matForward_dev; //R2C N->N/2+1
	matForward_dev = NULL;
	hipMalloc((void**)&matForward_dev, sizeof(hipfftComplex) * matSize); //R2C N->N/2+1


	hipfftHandle plan;
	/*
	��άFFT�㷨ʵ���У�ͬһάFFT��ͬ���ǣ�
	(1) ���������û��BATCH��������NY��NXΪ������NYΪ������
	(2) FFT�����任������λ�ú����λ�ò�ͬ��*/
	hipfftPlan2d(&plan, 4, 4, HIPFFT_C2C);  //  �� ��
	hipfftExecC2C(plan, (hipfftComplex*)mat_dev, (hipfftComplex*)matForward_dev,HIPFFT_FORWARD);
	hipMemcpy((hipfftComplex*)matForward, (hipfftComplex*)matForward_dev, sizeof(hipfftComplex) * matSize, hipMemcpyDeviceToHost);
	for (int i = 0; i < matSize; ++i)
	{
		//if (i == 0) cout << endl;
		//if (i % 4== 0) cout << endl;
		cout << (matForward + i)->x << " " << (matForward + i)->y << endl;
	}

	//��任
	//hipfftHandle planInv;
	//hipfftPlan1d(&planInv,11, HIPFFT_C2R,1);

	hipFree(mat_dev);
	hipFree(matForward_dev);
	free(mat);
	free(matForward);

}