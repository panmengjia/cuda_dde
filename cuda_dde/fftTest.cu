#include "main.h"

//c2c c2r r2c https://blog.csdn.net/qq_17239003/article/details/79090803

void fft1d()
{
	const int signalLength = 15;
	hipfftComplex* signal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * signalLength);
	for (int i = 0; i < signalLength; i++)
	{
		(signal+i)->x = (float)(i + 1);
		(signal+i)->y = 0;
	}
	hipfftComplex* signalDev;
	hipMalloc((void**)&signalDev, sizeof(hipfftComplex) * signalLength);
	hipMemcpy(signalDev, signal, sizeof(hipfftComplex) * signalLength, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan1d(&plan, signalLength, HIPFFT_C2C,1);
	hipfftExecC2C(plan, (hipfftComplex*)signalDev, (hipfftComplex*)signalDev, HIPFFT_FORWARD);
	hipDeviceSynchronize();//wait to be done
	hipMemcpy(signal, signalDev, signalLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < signalLength; i++)
	{
		cout << (signal+i)->x << " " << (signal+i)->y << endl;
	}

	//��任
	hipfftComplex* signalInverse = (hipfftComplex*)malloc(sizeof(hipfftComplex) * signalLength);
	hipfftComplex* signalInvDev;
	hipMalloc((void**)&signalInvDev, sizeof(hipfftComplex) * signalLength);
	for (int i = 0; i < signalLength; i++)  //ȫ����ʼ��Ϊ0������y in-place������д������ݣ�Ҳ���Բ�����y�е�����
	{
		(signal + i)->x = 0;
		(signal + i)->y = 0;
	}
	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}


	hipfftHandle planInv;
	hipfftPlan1d(&plan, signalLength, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, (hipfftComplex*)signalDev, signalInvDev, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	hipMemcpy(signalInverse, signalInvDev, signalLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}

	hipfftDestroy(plan);
	free(signal);
	hipFree(signalDev);

}