#include "hip/hip_runtime.h"
#include "main.h"

//c2c c2r r2c https://blog.csdn.net/qq_17239003/article/details/79090803
//�ܺõĲ������£�https://zhuanlan.zhihu.com/p/34587739


__global__ void kernel_normalize1d(hipfftComplex* signal,const int signalLength)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < signalLength) //�Ǳ�����ʷǷ��ڴ���
	{
		(signal + idx)->x /= signalLength;
		(signal + idx)->y /= signalLength;
	}
}

void fft1d()
{
	// Use cuda event to catch time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const int signalLength = 15;
	hipfftComplex* signal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * signalLength);
	for (int i = 0; i < signalLength; i++)
	{
		(signal+i)->x = (float)(i + 1);
		(signal+i)->y = 0;
	}
	hipfftComplex* signalDev;
	hipMalloc((void**)&signalDev, sizeof(hipfftComplex) * signalLength);
	hipMemcpy(signalDev, signal, sizeof(hipfftComplex) * signalLength, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan1d(&plan, signalLength, HIPFFT_C2C,1);
	hipfftExecC2C(plan, (hipfftComplex*)signalDev, (hipfftComplex*)signalDev, HIPFFT_FORWARD);
	hipDeviceSynchronize();//wait to be done
	hipMemcpy(signal, signalDev, signalLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < signalLength; i++)
	{
		cout << (signal+i)->x << " " << (signal+i)->y << endl;
	}

	//��任
	hipfftComplex* signalInverse = (hipfftComplex*)malloc(sizeof(hipfftComplex) * signalLength);
	hipfftComplex* signalInvDev;
	hipMalloc((void**)&signalInvDev, sizeof(hipfftComplex) * signalLength);
	for (int i = 0; i < signalLength; i++)  //ȫ����ʼ��Ϊ0������y in-place������д������ݣ�Ҳ���Բ�����y�е�����
	{
		(signal + i)->x = 0;
		(signal + i)->y = 0;
	}
	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}


	hipfftHandle planInv;
	hipfftPlan1d(&plan, signalLength, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, (hipfftComplex*)signalDev, signalInvDev, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	hipMemcpy(signalInverse, signalInvDev, signalLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}
	//�˹�һ������
	//���Բ������йܷ�ʽ
	const unsigned int blockX = 32;
	// Specify block size
	const dim3 block(blockX, 1);
	const dim3 grid(1);


	// Start time
	hipEventRecord(start, 0);  //��ʱ����
	kernel_normalize1d <<< grid, block >>> (signalInvDev,signalLength);//����ʵ���������߳�����32
	hipMemcpy(signalInverse, signalInvDev, signalLength * sizeof(hipfftComplex),hipMemcpyDeviceToHost);


	for (int i = 0; i < signalLength; i++)
	{
		cout << (signalInverse + i)->x << " " << (signalInverse + i)->y << endl;
	}

	hipfftDestroy(plan);
	free(signal);
	hipFree(signalDev);

}


vector<vector<float>> extractConvMatfft()
{
	const string& str = "C:/Users/b515/Desktop/pmj/cuda_dde/fenglin/15/";
	vector<vector<float>> HVSFT;
	HVSFT.resize(15);
	//    unsigned int counter = 0;
	for (int i = 1; i < 16; i++)
	{
		ifstream dataFile(str + to_string(i) + ".txt");
		float dataElement;
		while (dataFile >> dataElement)
		{
			HVSFT[i - 1].push_back(dataElement);
		}
		dataFile.close();
	}
	return HVSFT;
}

__global__ void kernel_normalize2d()
{

}

//�ù��̶�������ά����������
//����ָ����Բ���
void fft2d()
{
	vector<vector<float>> dataArray = extractConvMatfft();
	//for (int i = 0;  i < 15 * 15; i++)
	//{
	//	cout << *(dataArray[0].data() +i) << "  ";  //dataArray[0][i]  *(dataArray[0].data() +i)��Ȼ����Ҫǿ������ת��
	//}
	const unsigned int matSize = 20;
	hipfftReal (*mat)[5];
	mat = (hipfftReal(*)[5])malloc(sizeof(hipfftReal) * matSize);  //4row 5col
	for (int i = 0; i < 20; i++)
	{
		//*(*(mat + i/5)+i%5) = i + 1;
		*(*mat + i) = i + 1; //��ָ������϶�������е�ַ��ָ�������ֵ���ǵ�ַ
		//cout << mat[i / 5][i % 5] << endl;
	}
	hipfftReal(*mat_dev)[5];
	mat_dev = NULL;
	hipMalloc((void**)mat_dev, sizeof(hipfftReal [5]) * matSize/5); //hipfftReal
	hipMemcpy(*mat_dev, *mat, sizeof(hipfftReal) * matSize, hipMemcpyHostToDevice);
	hipfftComplex(*matForward)[5];
	matForward = (hipfftComplex(*)[5])malloc(sizeof(hipfftReal [5]) * matSize/5);  //�Ե�ַ����ǿ������ת����Ӧ�ö��Ƕ���ֵǿ������ת��
	hipfftComplex(*matForward_dev)[5];
	matForward_dev = NULL;
	hipMalloc((void**)matForward_dev, sizeof(hipfftComplex) * matSize);
	//matForward_dev = (hipfftComplex(*)[5])malloc(sizeof(hipfftComplex) * matSize);  //�Ե�ַ����ǿ������ת����Ӧ�ö��Ƕ���ֵǿ������ת��


	hipfftHandle plan;
	/*
	��άFFT�㷨ʵ���У�ͬһάFFT��ͬ���ǣ�
    (1) ���������û��BATCH��������NY��NXΪ������NYΪ������
    (2) FFT�����任������λ�ú����λ�ò�ͬ��*/
	hipfftPlan2d(&plan, 4, 5, HIPFFT_R2C);  //  �� ��
	hipfftExecR2C(plan,(hipfftReal*)*mat_dev, (hipfftComplex*)*matForward_dev);
	hipMemcpy((hipfftComplex*)*matForward, (hipfftComplex*)*matForward_dev, sizeof(hipfftComplex) * matSize, hipMemcpyDeviceToHost);
	for (int i = 0; i < matSize; ++i)
	{
		cout << (*matForward + i)->x << " " << (*matForward + i)->y << endl;
	}


	hipFree(*mat_dev);
	hipFree(*matForward_dev);
	free(*mat);
	
}

void fft3d()
{
	vector<vector<float>> dataArray = extractConvMatfft();
	//for (int i = 0;  i < 15 * 15; i++)
	//{
	//	cout << *(dataArray[0].data() +i) << "  ";  //dataArray[0][i]  *(dataArray[0].data() +i)��Ȼ����Ҫǿ������ת��
	//}
	const unsigned int matSize = 16;
	hipfftComplex* mat;
	mat = (hipfftComplex*)malloc(sizeof(hipfftComplex) * matSize);  //5row 4col
	for (int i = 0; i < matSize; i++)
	{
		(*(mat + i)).x = (i + 1); //��ָ������϶�������е�ַ��ָ�������ֵ���ǵ�ַ
		(mat + i)->y = 0;
		cout << mat[i].x << endl;
	}
	hipfftComplex*  mat_dev;  //N
	mat_dev = NULL;
	hipMalloc((void**)&mat_dev, sizeof(hipfftComplex) * matSize);  // //hipfftComplex
	hipMemcpy(mat_dev, mat, sizeof(hipfftComplex) * matSize, hipMemcpyHostToDevice); 
	hipfftComplex* matForward; //R2C N->N/2+1
	matForward = (hipfftComplex*)malloc(sizeof(hipfftComplex) * matSize);  //�Ե�ַ����ǿ������ת����Ӧ�ö��Ƕ���ֵǿ������ת��
	hipfftComplex* matForward_dev; //R2C N->N/2+1
	matForward_dev = NULL;
	hipMalloc((void**)&matForward_dev, sizeof(hipfftComplex) * matSize); //R2C N->N/2+1


	hipfftHandle plan;
	/*
	��άFFT�㷨ʵ���У�ͬһάFFT��ͬ���ǣ�
	(1) ���������û��BATCH��������NY��NXΪ������NYΪ������
	(2) FFT�����任������λ�ú����λ�ò�ͬ��*/
	hipfftPlan2d(&plan, 4, 4, HIPFFT_C2C);  //  �� ��
	hipfftExecC2C(plan, (hipfftComplex*)mat_dev, (hipfftComplex*)matForward_dev,HIPFFT_FORWARD);
	hipMemcpy((hipfftComplex*)matForward, (hipfftComplex*)matForward_dev, sizeof(hipfftComplex) * matSize, hipMemcpyDeviceToHost);
	for (int i = 0; i < matSize; ++i)
	{
		//if (i == 0) cout << endl;
		//if (i % 4== 0) cout << endl;
		cout << (matForward + i)->x << " " << (matForward + i)->y << endl;
	}

	//��任
	//hipfftHandle planInv;
	//hipfftPlan1d(&planInv,11, HIPFFT_C2R,1);

	hipFree(mat_dev);
	hipFree(matForward_dev);
	free(mat);
	free(matForward);

}

////////////////////////////////////////////////////////cuda dft mulSpectrum////////////////////////////////////////////////////////////////////

/*
�˺����ڷ����ڴ�ʱ���Զ���������
Ƶ�����֮�����任��ͼ��ᱻ��ȡ����ͬ��λ��

*/



//Nvidia cufft https://docs.nvidia.com/cuda/cufft/index.html#cufft-setup
/// <summary>
/// �Ժ˺�������Ҷ�任��kernel  real (M,N)->complex (M,N/2+1)
/// 
/// 2D	C2C	  (N1,N2)     hipfftComplex	(N1,N2)     hipfftComplex
/// 2D	C2R	  (N1,N2/2+1) hipfftComplex	(N1,N2)     hipfftReal
/// 2D	R2C	  (N1,N2)     hipfftReal	    (N1,N2/2+1) hipfftComplex
///  N1,nx <=> h; N2,ny <=> w
/// 
/// </summary>
/// <param name="indata"></�任֮ǰ��ʵ�����ݣ�h*w*sizeof(hipfftReal) bytes>
/// <param name="outdata"></�任֮��ĸ������ݣ�h*(w/2+1)*sizeof(hipfftComplex) bytes>
/// <param name="kh"></ height of image(or kernel),both have the same size>
/// <param name="kw"></weight of image(or kernel)>

void fftKernel(hipfftReal* indata, hipfftComplex* outdata, const unsigned int kh, const unsigned int kw)
{
	//imgh == kh
	//imgw == kw
	hipfftReal* indata_dev;
	hipfftComplex* outdata_dev;
	hipMalloc((void**)&indata_dev, sizeof(hipfftReal) * kh * kw);
	hipMalloc((void**)&outdata_dev, sizeof(hipfftComplex) * kh * (kw / 2 + 1));
	hipMemcpy(indata_dev, indata, sizeof(hipfftReal) * kw * kh, hipMemcpyHostToDevice);

	hipfftHandle planForward;
	hipfftPlan2d(&planForward, kh, kw, HIPFFT_R2C);
	hipfftExecR2C(planForward, indata_dev, outdata_dev);
	hipDeviceSynchronize();

	hipMemcpy(outdata, outdata_dev, sizeof(hipfftComplex) * kh * (kw / 2 + 1), hipMemcpyDeviceToHost);

	hipfftDestroy(planForward);
	hipFree(outdata_dev);
	hipFree(indata_dev);
}


/// <summary>
/// Ƶ����˵ĺ˺������������ ���������
/// </summary>
/// <param name="imgdata_dev"></���豸�ϵ�ͼ������>
/// <param name="outdata_dev"></����out-of-place�洢���������ݣ����������Ⱦ>
/// <param name="kdata_dev"></�����ģ�壬����ʼ�շ����豸��>
/// <param name="imgh"></height of image>
/// <param name="imgw"></width of image>
/// <returns></returns>

__global__ void kernel_convfft(const hipfftComplex* imgdata_dev, hipfftComplex* outdata_dev, const hipfftComplex* kdata_dev, const unsigned int imgh, const unsigned int imgw)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	//    if(x< imgw && y < imgh/2 )
	//    {
	//        imgdata_dev[y*imgw + x].x = imgdata_dev[y*imgw + x].x * kdata_dev[y*imgw + x].x;
	//        imgdata_dev[y*imgw + x].y = imgdata_dev[y*imgw + x].y * kdata_dev[y*imgw + x].y;
	//    }
	//    else if(y >= imgh/2 && x <imgw && y <imgh)
	//    {
	//        imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].x = imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].x * kdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].x;
	//        imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].y = imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].y * kdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].y;
	//    }
	//if (x == 0 || y == 0)
	//{
	//	imgdata_dev[y * imgw + x].x = imgdata_dev[y * imgw + x].x * kdata_dev[y * imgw + x].x - imgdata_dev[y * imgw + x].y * kdata_dev[y * imgw + x].y;
	//	imgdata_dev[y * imgw + x].y = imgdata_dev[y * imgw + x].y * kdata_dev[y * imgw + x].x + imgdata_dev[y * imgw + x].x * kdata_dev[y * imgw + x].y;
	//}
	//else if (y > 0 && x > 0 && x < (imgw / 2 + 1))
	//{
	//	imgdata_dev[y * imgw + x].x = imgdata_dev[y * imgw + x].x * kdata_dev[y * imgw + x].x - imgdata_dev[y * imgw + x].y * kdata_dev[y * imgw + x].y;
	//	imgdata_dev[y * imgw + x].y = imgdata_dev[y * imgw + x].y * kdata_dev[y * imgw + x].x + imgdata_dev[y * imgw + x].x * kdata_dev[y * imgw + x].y;
	//}
	unsigned int h = imgh;
	unsigned int w = imgw / 2 + 1;
	if (x < w && y < h) //�豸�Ϸ��ʷǷ��ڴ���̻߳ᱻֹͣ
	{
		//float ri = imgdata_dev[y * w + x].x;
		//float ii = imgdata_dev[y * w + x].y;
		//float rk = kdata_dev[y * w + x].x;
		//float ik = kdata_dev[y * w + x].y;  //�˵�������λ

		//if ()
		//{
			outdata_dev[y * w + x].x = imgdata_dev[y * w + x].x * kdata_dev[y * w + x].x - imgdata_dev[y * w + x].y * kdata_dev[y * w + x].y; //ԭ��д��imgw
			outdata_dev[y * w + x].y = imgdata_dev[y * w + x].y * kdata_dev[y * w + x].x + imgdata_dev[y * w + x].x * kdata_dev[y * w + x].y;
		//}
		//else if (x == 0 && y == 0)
		//{
		//	imgdata_dev[y * w + x].x = imgdata_dev[y * w + x].x * kdata_dev[y * w + x].x;
		//}
	}
}

bool first1 = true;
hipfftReal* imgdata_dev;
hipfftReal* outdata_dev;
hipfftComplex* outdataComplex_dev;
hipfftComplex* imgdataComplex_dev;
hipfftComplex* kdata_dev;
hipfftHandle planForward, planInverse;

void fftImgKernel(const hipfftReal* imgdata, hipfftReal* outdata, const hipfftComplex* kdata, const unsigned int imgh, const unsigned int imgw)
{
	if (first1)
	{
		//img
		hipMalloc((void**)&imgdata_dev, sizeof(hipfftReal) * imgh * imgw);
		hipMalloc((void**)&outdata_dev, sizeof(hipfftReal) * imgh * imgw);
		//img complex
		hipMalloc((void**)&imgdataComplex_dev, sizeof(hipfftComplex) * imgh * (imgw / 2 + 1));//����Ҫ���룬ǰ����ܾ������ݴ���
		hipMalloc((void**)&outdataComplex_dev, sizeof(hipfftComplex) * imgh * (imgw / 2 + 1));
		//kernel
		hipMalloc((void**)&kdata_dev, sizeof(hipfftComplex) * imgh * (imgw / 2 + 1));
		//handle
		hipfftPlan2d(&planForward, imgh, imgw, HIPFFT_R2C);
		hipfftPlan2d(&planInverse, imgh, imgw, HIPFFT_C2R);
		first1 = false;
	}
    //memory copy kernel and image from host to device
	hipMemcpy(kdata_dev, kdata, sizeof(hipfftComplex) * imgh * (imgw / 2 + 1), hipMemcpyHostToDevice);
	hipMemcpy(imgdata_dev, imgdata, sizeof(hipfftReal) * imgh * imgw , hipMemcpyHostToDevice);
	//fft forward transform
	hipfftExecR2C(planForward, imgdata_dev, imgdataComplex_dev);
	hipDeviceSynchronize();
	//multiply kernel and image
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((imgw/2 + 1 + block.x - 1) / block.x, (imgh + block.y - 1) / block.y);
	kernel_convfft << <grid, block >> > (imgdataComplex_dev, outdataComplex_dev, kdata_dev, imgh, imgw);
	//fft inverse transform
	hipfftExecC2R(planInverse, outdataComplex_dev, outdata_dev);
	hipDeviceSynchronize();
	hipMemcpy(outdata, outdata_dev, sizeof(hipfftReal) * imgh * imgw, hipMemcpyDeviceToHost);
	//Mat outImg = Mat(imgh, imgw, CV_32F, outdata);
	//cout << outImg(Rect(0, 0, 15, 15)) << endl;
	//    hipFree(imgdata_dev);
	//    hipFree(imgdataComplex_dev);
	//    hipFree(kdata_dev);
	//    hipfftDestroy(planForward);
	//    hipfftDestroy(planInverse);
}


void ycbcrUpdate1119(const Mat& IM_result_cbcr, const Mat& IM_bri_T, Mat& IM_result_cbcr_re);

vector<vector<float>> extractConvMat01119();

int testcudamulSpectrum()
{
	VideoCapture* cap = new VideoCapture("C:\\Users\\b515\\Desktop\\pmj\\cuda_dde\\fenglin\\130 (3).avi");
	if (!cap->isOpened())
	{
		cout << "video is empty!!" << endl;
	}
	Mat frame; //1080 1920
	//discard the first 25 frames
	while (cap->read(frame))
	{
		static unsigned int counter = 0;
		if (++counter == 25)
		{
			break;
		}
	}
	Mat meanFrame, stdDevFrame, singleChannel/*IM_bri*/;
	double mean;
	vector<Mat> splitFrame2bgr3;

	Mat meanBri, stdDevBri;
	double thresholdRate;

	Mat IM_bri_T0;
	//
	vector<vector<float>> temp = extractConvMat01119();
	
	for (int i = 0; i < 15 * 15; i++)
	{
		if (i % 15 == 0) cout << endl;
		cout << temp[6][i] << " ";
	}

	vector<Mat> hsfatMat;
	for (int i = 0; i < 15; i++)
	{
		hsfatMat.push_back(Mat(FILTER_HEIGHT, FILTER_WIDTH, CV_32FC1, temp[i].data()));
	}
	cout << endl;
	cout << hsfatMat[6] << endl;
	cap->read(frame);
	int fM = getOptimalDFTSize(frame.rows);                               // ������DFT�ߴ磬Ϊ2�Ĵη�
	int fN = getOptimalDFTSize(frame.cols);

	hipfftComplex* kernelComplex[15];
	Mat kernelpadded[15];
	for (int i = 0; i < 15; ++i)
	{
		kernelpadded[i] = Mat(fM, fN, CV_32FC1);
		kernelComplex[i] = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fM * (fN / 2 + 1));
		//        copyMakeBorder(hsfatMat[i],kernelpadded[i], fM/2 - hsfatMat[i].rows/2-1, fM/2 - hsfatMat[i].rows/2, fN/2 - hsfatMat[i].cols/2-1, fN/2 - hsfatMat[i].cols/2, BORDER_CONSTANT, Scalar::all(0));   // opencv�еı߽���չ�������ṩ���ַ�ʽ��չ
		copyMakeBorder(hsfatMat[i], kernelpadded[i], 0, fM - hsfatMat[i].rows, 0, fN - hsfatMat[i].cols, BORDER_CONSTANT, Scalar::all(0));   // opencv�еı߽���չ�������ṩ���ַ�ʽ��չ
		fftKernel((hipfftReal*)kernelpadded[i].data, kernelComplex[i], kernelpadded[i].rows, kernelpadded[i].cols);
	}

	hipfftComplex* kernelComplex6 = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fM * (fN / 2 + 1));
	Mat kernelpadded6;
	copyMakeBorder(hsfatMat[6], kernelpadded6, 0, fM - hsfatMat[6].rows, 0, fN - hsfatMat[6].cols, BORDER_CONSTANT, Scalar::all(0));   // opencv�еı߽���չ�������ṩ���ַ�ʽ��չ
	fftKernel((hipfftReal*)kernelpadded6.data, kernelComplex6, kernelpadded6.rows, kernelpadded6.cols);
	//cout << kernelpadded6 << endl;
	cout << hsfatMat[6] << endl;

	//for (int i = 0; i < fM; i++)
	//{
	//	for (int j = 0; j < (fN / 2 + 1); j++)
	//	{
	//		cout <<kernelComplex6[i * (fN / 2 + 1) + j].x << endl;
	//	}
	//}
	Mat kernelComplex6Mat = Mat(fM, (fN / 2 + 1), CV_32FC2, kernelComplex6);
	cout << kernelComplex6Mat(Rect(0,0,15,15)) << endl;

	Mat planes[] = { kernelpadded6,Mat::zeros(kernelpadded6.size(),CV_32F) };
	Mat complexImg;
	merge(planes, 2, complexImg);
	dft(complexImg, complexImg);



	while (cap->read(frame))  //type CV_8UC3
	{
		double time = (double)getTickCount();

		meanStdDev(frame, meanFrame, stdDevFrame);
		mean /*results_de_mean*/ = (meanFrame.at<Vec3d>(0, 0)[0] + meanFrame.at<Vec3d>(0, 0)[1] + meanFrame.at<Vec3d>(0, 0)[2]) / 3;
		split(frame, splitFrame2bgr3);
		singleChannel = 0.257 * splitFrame2bgr3[2] + 0.564 * splitFrame2bgr3[1] + 0.098 * splitFrame2bgr3[0] + 0.0;
		meanStdDev(singleChannel, meanBri, stdDevBri);
		thresholdRate = meanBri.at<double>(0, 0) / stdDevBri.at<double>(0, 0) * meanBri.at<double>(0, 0) / 80;
		thresholdRate += 0.2;
		if (thresholdRate > 0.6)
		{
			thresholdRate = 0.6;
		}
		cout << "thresholdRate*10:   " << thresholdRate * 10 << endl;
		Mat H_S_f_A_T = hsfatMat[(unsigned int)(thresholdRate * 10)];

		IM_bri_T0 = singleChannel;

		Mat IM_bri_T = Mat::zeros(IM_bri_T0.size(), IM_bri_T0.type());
		//        imshow("IM_bri_T",IM_bri_T0);

		int M = getOptimalDFTSize(IM_bri_T.rows);                               // ������DFT�ߴ磬Ϊ2�Ĵη�
		int N = getOptimalDFTSize(IM_bri_T.cols);

		Mat padded;
		copyMakeBorder(singleChannel, padded, 0, M - IM_bri_T0.rows, 0, N - IM_bri_T0.cols, BORDER_CONSTANT, Scalar::all(0));   // opencv�еı߽���չ�������ṩ���ַ�ʽ��չ

//        Mat planes = Mat_<float>(padded);

//        imshow("planes",planes);
		padded.convertTo(padded, CV_32FC1, 1.0 / 255.0);
		//        imshow("padded",padded);

		Mat paddedInverse = Mat::zeros(padded.size(), CV_32FC1);
		//        fftQt1(planes.data,paddedInverse.data,planes.rows,planes.cols);
		fftImgKernel((hipfftReal*)padded.data, (hipfftReal*)paddedInverse.data, kernelComplex6/*[(unsigned int)(thresholdRate * 10)]*/, padded.rows, padded.cols);
		//imshow("paddedInverse", paddedInverse);
		//paddedInverse = padded;
		imshow("paddedInverse", paddedInverse);
		paddedInverse.convertTo(paddedInverse, CV_8UC1, 255/1920.0/1080.0);

		imshow("paddedInverse1", paddedInverse);

		//      laplacianFilter_GPU_wrapper(IM_bri_T0, IM_bri_T, H_S_f_A_T);

		//      filter2D(IM_bri_T0,IM_bri_T,CV_64FC1,H_S_f_A_T);

		//        Mat meanOfIM_bri_T, stdDevOfIM_bri_T;
		//        meanStdDev(IM_bri_T, meanOfIM_bri_T, stdDevOfIM_bri_T);
		//        IM_bri_T = IM_bri_T / meanOfIM_bri_T.at<double>(0, 0) * meanBri.at<double>(0, 0) / 1;

		//        Mat IM_result_cbcr;
		//        cvtColor(frame, IM_result_cbcr, COLOR_BGR2YCrCb);
		//        Mat IM_result_cbcr_re;
		//        ycbcrUpdate(IM_result_cbcr, IM_bri_T, IM_result_cbcr_re);

		Mat meanOfIM_bri_T, stdDevOfIM_bri_T;
		meanStdDev(paddedInverse, meanOfIM_bri_T, stdDevOfIM_bri_T);
		paddedInverse = paddedInverse / meanOfIM_bri_T.at<double>(0, 0) * meanBri.at<double>(0, 0) / 1;

		Mat IM_result_cbcr;
		cvtColor(frame, IM_result_cbcr, COLOR_BGR2YCrCb);
		Mat IM_result_cbcr_re;
		ycbcrUpdate1119(IM_result_cbcr, paddedInverse, IM_result_cbcr_re);

		imshow("frame", frame);
		imshow("IM_result_cbcr_re", IM_result_cbcr_re);
		waitKey(1);
		double fps = getTickFrequency() / (getTickCount() - time);
		cout << "fps" << fps << endl;
	}

	delete cap;
	delete[] kernelComplex;
	return 0;
}




void ycbcrUpdate1119(const Mat& IM_result_cbcr, const Mat& IM_bri_T, Mat& IM_result_cbcr_re)
{
	vector<Mat> channelsOfIM;
	split(IM_result_cbcr, channelsOfIM);
	Mat IM_bri_T_8U;
	IM_bri_T.convertTo(IM_bri_T_8U, CV_8UC1);
	channelsOfIM[0] = IM_bri_T_8U;
	merge(channelsOfIM, IM_result_cbcr_re);
	cvtColor(IM_result_cbcr_re, IM_result_cbcr_re, COLOR_YCrCb2BGR);
}

vector<vector<float>> extractConvMat01119()
{
	const string& str = "C:/Users/b515/Desktop/pmj/cuda_dde/fenglin/15/";
	vector<vector<float>> HVSFT;
	HVSFT.resize(15);
	//    unsigned int counter = 0;
	for (int i = 1; i < 16; i++)
	{
		ifstream dataFile(str + to_string(i) + ".txt");
		float dataElement;
		while (dataFile >> dataElement)
		{
			HVSFT[i - 1].push_back(dataElement);
		}
		dataFile.close();
	}
	return HVSFT;
}
////////////////////////////////////////////////////////cuda dft mulSpectrum////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////cuda dft mulSpectrum����////////////////////////////////////////////////////////////////////
/*
* cuda dft mulSpectrum
* 2020.11.19�̳�ǰ��ʹ��cuda�˺���д��Ƶ����ˣ��޸Ĵ��󣬹淶����
* 
* 
* 
*/

#define CVTCOLOR (true)

void cudaFFTmulSpectrum1119float()
{
	/////////////////////////////////////////////��ȡtxt////////////////////////////////////////////
	const string str = "C:/Users/b515/Desktop/pmj/cuda_dde/fenglin/" + to_string(FILTER_WIDTH) + "/";
	vector<vector<float>> dataAllFile;
	vector<float> dataPerFile;
	float dataElement;
	//vector<Mat> kernelMat;
	for (int i = 0; i < 15; ++i)
	{
		//vector<float> dataPerFile;
		ifstream dataFile(str + to_string(i + 1) + ".txt");
		while (dataFile >> dataElement)
		{
			dataPerFile.push_back(dataElement);
		}
		dataFile.close();
		dataAllFile.push_back(dataPerFile);
		dataPerFile.clear();
	}

	vector<Mat> kernelMat;
	hipfftComplex* kernelComplex[15];
	Mat kernelPadded;

	for (int i = 0; i < 15; ++i) //������ֻ�ܵõ����һ�ε����ݽ����֮ǰ�Ķ���������
	{
		kernelComplex[i] = (hipfftComplex*)malloc(sizeof(hipfftComplex) * IMG_HEIGHT * (IMG_WIDTH / 2 + 1));
		kernelMat.push_back(Mat(FILTER_HEIGHT, FILTER_WIDTH, CV_32FC1, dataAllFile[i].data()));
		copyMakeBorder(kernelMat[i], kernelPadded, 0, IMG_HEIGHT - kernelMat[i].rows, 0, IMG_WIDTH - kernelMat[i].cols, BORDER_CONSTANT, Scalar(0)); //Scalar::all(0)
		fftKernel((hipfftReal*)kernelPadded.data, kernelComplex[i], kernelPadded.rows, kernelPadded.cols);/*IMG_HEIGHT, IMG_WIDTH);*/
		//kernelMat[i].convertTo(kernelMat[i], CV_16FC1);
	}

	cout << kernelMat[6] << endl;
	//���Դ���
	//����fftKernel��dft�������ͬ������ȫһ����
	//opencv dft��complex��ʽ��cufft��cudaComplex�����ݽṹ��һ���� 
	//�����ȴ洢������
	hipfftComplex* kernelComplex6 = (hipfftComplex*)malloc(sizeof(hipfftComplex) * IMG_HEIGHT * (IMG_WIDTH / 2 + 1));
	Mat kernelPadded6;
	copyMakeBorder(kernelMat[6], kernelPadded6, 0, IMG_HEIGHT - kernelMat[6].rows, 0, IMG_WIDTH - kernelMat[6].cols, BORDER_CONSTANT, Scalar::all(0));   // opencv�еı߽���չ�������ṩ���ַ�ʽ��չ
	fftKernel((hipfftReal*)kernelPadded6.data, kernelComplex6, kernelPadded6.rows, kernelPadded6.cols);
	//cout << kernelpadded6 << endl;
	cout << kernelMat[6] << endl;

	//ͨ�������洢�����߿���������Ӧ
	Mat kernelComplex6Mat = Mat(IMG_HEIGHT, (IMG_WIDTH / 2 + 1), CV_32FC2, kernelComplex6);
	cout << kernelComplex6Mat(Rect(0, 0, 15, 15)) << endl;

	Mat planes[] = { kernelPadded6,Mat::zeros(kernelPadded6.size(),CV_32F) };
	Mat complexImg;
	merge(planes, 2, complexImg);
	dft(complexImg, complexImg);
	/////////////////////////////////////////////��ȡtxt//////////////////////////////////////////// 

	/////////////////////////////////////////////DDEfilter////////////////////////////////////////////
	VideoCapture* cap = new VideoCapture("C:\\Users\\b515\\Desktop\\pmj\\cuda_dde\\fenglin\\130 (3).avi");
	if (!cap->isOpened())
	{
		cout << "video is empty!!" << endl;
	}
	Mat frame; //1080 1920
	Mat outFrame, singleFrame, meanSF, stdDevSF, selectedKernelMat, SFfp32, SFfp32out, frameYUV;
	vector<Mat> frame3channels, frame3YUV;
	double threshRate;

	while (1)
	{
		double time = (double)getTickCount();
		double time2 = (double)getTickCount();
		cap->read(frame);
#if CVTCOLOR
		cvtColor(frame, singleFrame, COLOR_BGR2GRAY);
#else   
		split(frame, frame3channels);
		singleFrame = 0.257 * frame3channels[2] + 0.564 * frame3channels[1] + 0.098 * frame3channels[0];
#endif
		meanStdDev(singleFrame, meanSF, stdDevSF);

		threshRate = meanSF.at<double>(0, 0) / stdDevSF.at<double>(0, 0) * meanSF.at<double>(0, 0) / 80 + 0.2;
		if (threshRate > 0.6)
		{
			threshRate = 0.6;
		}
		cout << "threshRate*10: " << threshRate * 10 << endl;
		selectedKernelMat = kernelMat[(unsigned __int64)(threshRate * 10)];
		time2 = (double)(getTickCount() - time2) * 1000 / getTickFrequency();
		cout << "   time while pre   =  " << time2 << "  ms" << endl;

		time2 = (double)getTickCount();
		//singleFrame.convertTo(SFfp16, CV_16FC1);
		//SFfp16out = Mat(SFfp16.size(), CV_16FC1); //�ѿ����ڴ棬���������������-23.0
		singleFrame.convertTo(SFfp32, CV_32FC1);
		SFfp32out = Mat(SFfp32.size(), CV_32FC1); //�ѿ����ڴ棬���������������-23.0

		time2 = (double)(getTickCount() - time2) * 1000 / getTickFrequency();
		cout << "   time creat SFfp32out convertTo SFfp32Frame   =  " << time2 << "  ms" << endl;

		time2 = (double)getTickCount();
		/* laplacianFilter_GPU_wrapper(SFfp16, SFfp16out, selectedKernelMat);*/
		 //convfp16((__half*)SFfp16.data, (__half*)SFfp16out.data, (__half*)selectedKernelMat.data, SFfp16.cols, SFfp16.rows); //.ptr() .data���صĶ���һ��ָ�루��ָ�룩

		//timeCall = (double)getTickCount();
		//convolveDFT(SFfp32, selectedKernelMat, SFfp32out);

		fftImgKernel((hipfftReal*)SFfp32.data, (hipfftReal*)SFfp32out.data, kernelComplex[(unsigned int)(threshRate * 10)], SFfp32.rows, SFfp32.cols);

		time2 = (double)(getTickCount() - time2) * 1000 / getTickFrequency();
		cout << "   time convDFT   =  " << time2 << "  ms" << endl;

		time2 = (double)getTickCount();
		//SFfp32out.convertTo(SFfp32out, CV_8UC1);
		SFfp32out.convertTo(SFfp32out, CV_8UC1,1/((float)IMG_HEIGHT*(float)IMG_WIDTH)); //���ǲ�������Ҫ��Ҫ����255
		time2 = (double)(getTickCount() - time2) * 1000 / getTickFrequency();
		cout << "   time convertTo SFfp32out fp32->8u  =  " << time2 << "  ms" << endl;

		time2 = (double)getTickCount();
		Scalar meanSFfp32out = mean(SFfp32out);
		SFfp32out = SFfp32out / meanSFfp32out[0] * meanSF.at<double>(0, 0) / 1;
		time2 = (double)(getTickCount() - time2) * 1000 / getTickFrequency();
		cout << "   time mean of SF32fp32out  =  " << time2 << "  ms" << endl;

		time2 = (double)getTickCount();
		cvtColor(frame, frameYUV, COLOR_BGR2YCrCb);
		split(frameYUV, frame3YUV);
		frame3YUV[0] = SFfp32out;
		merge(frame3YUV, frameYUV);
		cvtColor(frameYUV, outFrame, COLOR_YCrCb2BGR);
		time2 = (double)(getTickCount() - time2) * 1000 / getTickFrequency();
		cout << "   time cvtColor yuv  =  " << time2 << "  ms" << endl;

		time2 = (double)getTickCount();
		imshow("frame", frame);
		imshow("outFrame", outFrame);
		time2 = (double)(getTickCount() - time2) * 1000 / getTickFrequency();
		cout << "   time imshow  =  " << time2 << "  ms" << endl;


		double fps = getTickFrequency() / (getTickCount() - time);
		time = (double)(getTickCount() - time) * 1000 / getTickFrequency();
		cout << "  time total =  " << time << "  ms" << endl;
		cout << "fps" << fps << endl;
		waitKey(1);
	}

	delete cap;
	/////////////////////////////////////////////DDEfilter////////////////////////////////////////////
}

////////////////////////////////////////////////////////cuda dft mulSpectrum����////////////////////////////////////////////////////////////////////