#include "main.h"

//cufft_examples/src/fp16_common.hpp


void fftQt1(uchar* indata,uchar* outdata,const unsigned int heigth,const unsigned int width)
{
    hipfftComplex* dataComplex_dev;
    hipfftReal* indata_dev;
    hipfftReal* outdata_dev;
    hipMalloc((void**)&dataComplex_dev,sizeof(hipfftComplex)*heigth*(width/2+1));
    hipMalloc((void**)&indata_dev,sizeof(hipfftReal)*heigth*width);
     hipMalloc((void**)&outdata_dev,sizeof(hipfftReal)*heigth*width);
    hipMemcpy(indata_dev,indata,sizeof(hipfftReal)*heigth*width,hipMemcpyHostToDevice);
    hipfftHandle planForward,planInverse;

    hipfftPlan2d(&planForward,heigth,width,HIPFFT_R2C);
    hipfftPlan2d(&planInverse,heigth,width,HIPFFT_C2R);

    hipfftExecR2C(planForward,indata_dev,dataComplex_dev);
    hipDeviceSynchronize();
    hipfftExecC2R(planInverse,dataComplex_dev,outdata_dev);
    hipDeviceSynchronize();
    hipMemcpy(outdata,outdata_dev,sizeof(hipfftReal)*heigth*width,hipMemcpyDeviceToHost);

    hipfftDestroy(planForward);
    hipfftDestroy(planInverse);
    hipFree(dataComplex_dev);
    hipFree(indata_dev);
}


void mainfftQT1()
{
    Mat img = imread("/home/nvidia/Downloads/8700e1c96cbbdbfc8bb32a700fd8fc85.jpg",IMREAD_GRAYSCALE);
    imshow("img",img);
    if( img.empty() )
    {
        std::cout <<"Cannot read image file"<<std::endl;
    }

    int M = getOptimalDFTSize( img.rows );                               // 获得最佳DFT尺寸，为2的次方
    int N = getOptimalDFTSize( img.cols );
    Mat padded;
    copyMakeBorder(img, padded, 0, M - img.rows, 0, N - img.cols, BORDER_CONSTANT, Scalar::all(0));   // opencv中的边界扩展函数，提供多种方式扩展

    Mat planes = Mat_<float>(padded);

    Mat paddedInverse = Mat::zeros(planes.size(),CV_32FC1);
    fftQt1(planes.data,paddedInverse.data,planes.rows,planes.cols);
//   for(int i=0;i<planes.rows;++i)
//   {
//       for(int j=0;j<planes.cols;++j)
//       {
//          paddedInverse.at<float>(i,j) = paddedInverse.at<float>(i,j)/(planes.rows*planes.cols);
//       }
//   }
   paddedInverse.convertTo(paddedInverse,CV_8UC1,1.0/(float)(planes.rows*planes.cols));
   imshow("paddedInverse",paddedInverse);



}



