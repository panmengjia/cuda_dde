#include "hip/hip_runtime.h"
#include "main.h"

//cufft_examples/src/fp16_common.hpp
#include "string"





void fftKernel(void* indata,void* outdata,const unsigned int kh,const unsigned int kw)
{
    hipfftReal* indata_dev;
    hipfftComplex* outdata_dev;
    hipMalloc((void**)&indata_dev,sizeof(hipfftReal)*kh*kw);
    hipMalloc((void**)&outdata_dev,sizeof(hipfftComplex)*kh*(kw/2+1));
    hipMemcpy(indata_dev,indata,sizeof(hipfftReal)*kw*kh,hipMemcpyHostToDevice);

    hipfftHandle planForward;
    hipfftPlan2d(&planForward,kh,kw,HIPFFT_R2C);
    hipfftExecR2C(planForward,indata_dev,outdata_dev);
    hipDeviceSynchronize();

    hipMemcpy(outdata,outdata_dev,sizeof(hipfftReal)*kh*kw,hipMemcpyDeviceToHost);

    hipfftDestroy(planForward);
    hipFree(outdata_dev);
    hipFree(indata_dev);
}




void fftQt1(void* indata,void* outdata,const unsigned int heigth,const unsigned int width)
{
    hipfftComplex* dataComplex_dev;
    hipfftReal* indata_dev;
    hipfftReal* outdata_dev;
    hipMalloc((void**)&dataComplex_dev,sizeof(hipfftComplex)*heigth*(width/2+1));
    hipMalloc((void**)&indata_dev,sizeof(hipfftReal)*heigth*width);
    hipMalloc((void**)&outdata_dev,sizeof(hipfftReal)*heigth*width);
    hipMemcpy(indata_dev,indata,sizeof(hipfftReal)*heigth*width,hipMemcpyHostToDevice);
    hipfftHandle planForward,planInverse;

    hipfftPlan2d(&planForward,heigth,width,HIPFFT_R2C);
    hipfftPlan2d(&planInverse,heigth,width,HIPFFT_C2R);

    hipfftExecR2C(planForward,indata_dev,dataComplex_dev);
    hipDeviceSynchronize();
    hipfftExecC2R(planInverse,dataComplex_dev,outdata_dev);
    hipDeviceSynchronize();
    hipMemcpy(outdata,outdata_dev,sizeof(hipfftReal)*heigth*width,hipMemcpyDeviceToHost);

    hipfftDestroy(planForward);
    hipfftDestroy(planInverse);
    hipFree(dataComplex_dev);
    hipFree(indata_dev);
}

void mainfftQT1()
{
    Mat img = imread("/home/nvidia/Downloads/8700e1c96cbbdbfc8bb32a700fd8fc85.jpg",IMREAD_GRAYSCALE);
    imshow("img",img);
    if( img.empty() )
    {
        std::cout <<"Cannot read image file"<<std::endl;
    }

    int M = getOptimalDFTSize( img.rows );                               // 获得最佳DFT尺寸，为2的次方
    int N = getOptimalDFTSize( img.cols );
    Mat padded;
    copyMakeBorder(img, padded, 0, M - img.rows, 0, N - img.cols, BORDER_CONSTANT, Scalar::all(0));   // opencv中的边界扩展函数，提供多种方式扩展

    Mat planes = Mat_<float>(padded);

    Mat paddedInverse = Mat::zeros(planes.size(),CV_32FC1);
    fftQt1(planes.data,paddedInverse.data,planes.rows,planes.cols);
//   for(int i=0;i<planes.rows;++i)
//   {
//       for(int j=0;j<planes.cols;++j)
//       {
//          paddedInverse.at<float>(i,j) = paddedInverse.at<float>(i,j)/(planes.rows*planes.cols);
//       }
//   }
   paddedInverse.convertTo(paddedInverse,CV_8UC1,1.0/(float)(planes.rows*planes.cols));
   imshow("paddedInverse",paddedInverse);




}



#define BLOCKIZE 32
//@

__global__ void kernel_convfft(hipfftComplex* imgdata_dev,hipfftComplex* kdata_dev,const unsigned int imgh,const unsigned int imgw)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x< imgw && y < imgh/2 )
    {
        imgdata_dev[y*imgw + x].x = imgdata_dev[y*imgw + x].x * kdata_dev[y*imgw + x].x;
        imgdata_dev[y*imgw + x].y = imgdata_dev[y*imgw + x].y * kdata_dev[y*imgw + x].y;
    }
    else if(y >= imgh/2 && x <imgw && y <imgh)
    {
        imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].x = imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].x * kdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].x;
        imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].y = imgdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].y * kdata_dev[(imgh-1-y)*imgw + (imgw-1-x)].y;
    }

}

bool first =true;
hipfftReal* imgdata_dev;
hipfftComplex* imgdataComplex_dev;
hipfftComplex* kdata_dev;
hipfftHandle planForward,planInverse;

void fftImgKernel(void* imgdata,void* kdata,const unsigned int imgh,const unsigned int imgw)
{
    if(first)
    {
        //img

        hipMalloc((void**)&imgdata_dev,sizeof(hipfftReal)*imgh*imgw);


        //img complex

        hipMalloc((void**)&imgdataComplex_dev,sizeof(hipfftComplex)*imgh*imgw);

        //kernel

        hipMalloc((void**)&kdata_dev,sizeof(hipfftComplex)*imgh*imgw);



        //handle

        hipfftPlan2d(&planForward,imgh,imgw,HIPFFT_R2C);
        hipfftPlan2d(&planInverse,imgh,imgw,HIPFFT_C2R);
        first =false;
    }

    hipMemcpy(kdata_dev,kdata,sizeof(hipfftComplex)*imgh*imgw,hipMemcpyHostToDevice);
    hipMemcpy(imgdata_dev,imgdata,sizeof(hipfftReal)*imgh*imgw,hipMemcpyHostToDevice);


    //fft forward exec
    hipfftExecR2C(planForward,imgdata_dev,imgdataComplex_dev);
    hipDeviceSynchronize();

    //multiply kernel and image
    dim3 block(BLOCKIZE,BLOCKIZE);
    dim3 grid((imgw + block.x - 1) / block.x, (imgh + block.y - 1) / block.y);
    kernel_convfft<<<grid,block>>>(imgdataComplex_dev,kdata_dev,imgh,imgw);

    hipfftExecC2R(planInverse,imgdataComplex_dev,imgdata_dev);
    hipMemcpy(imgdata,imgdata_dev,sizeof(hipfftReal)*imgh*imgw,hipMemcpyDeviceToHost);

//    hipFree(imgdata_dev);
//    hipFree(imgdataComplex_dev);
//    hipFree(kdata_dev);
//    hipfftDestroy(planForward);
//    hipfftDestroy(planInverse);
}
